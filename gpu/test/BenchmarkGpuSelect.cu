#include "hip/hip_runtime.h"

/**
 * Copyright (c) 2015-present, Facebook, Inc.
 * All rights reserved.
 *
 * This source code is licensed under the CC-by-NC license found in the
 * LICENSE file in the root directory of this source tree.
 */

// Copyright 2004-present Facebook. All Rights Reserved.

#include "../utils/DeviceUtils.h"
#include "../utils/BlockSelectKernel.cuh"
#include "../utils/WarpSelectKernel.cuh"
#include "../utils/HostTensor.cuh"
#include "../utils/Timer.h"
#include "../utils/DeviceTensor.cuh"
#include "../test/TestUtils.h"
#include <algorithm>
#include <gtest/gtest.h>
#include <sstream>
#include <unordered_map>
#include <vector>
#include <hip/hip_runtime_api.h>
#include "/usr/local/google/home/blackhc/git/benchmark/src/stat.h"

int main(int argc, char** argv) {
  testing::InitGoogleTest(&argc, argv);
  return RUN_ALL_TESTS();
}

benchmark::Stat1_f testForSize(int batch_size, int n, int k, bool dir, bool warp) {
  benchmark::Stat1_f stats;
  faiss::gpu::CpuTimer timer;

  std::vector<float> v = faiss::gpu::randVecs(batch_size, n);
  faiss::gpu::HostTensor<float, 2, true> hostVal({batch_size, n});

  for (int r = 0; r < batch_size; ++r) {
    for (int c = 0; c < n; ++c) {
      hostVal[r][c] = v[r * n + c];
    }
  }

  // Select top-k on GPU
  faiss::gpu::DeviceTensor<float, 2, true> gpuVal(hostVal, 0);
  faiss::gpu::DeviceTensor<float, 2, true> gpuOutVal({batch_size, k});
  faiss::gpu::DeviceTensor<int, 2, true> gpuOutInd({batch_size, k});

  for (int i = 0 ; i < 20 ; i++) {
    hipDeviceSynchronize();
    float start_time = timer.elapsedMilliseconds();
    if (warp) {
      faiss::gpu::runWarpSelect(gpuVal, gpuOutVal, gpuOutInd, dir, k, 0);
    } else {
      faiss::gpu::runBlockSelect(gpuVal, gpuOutVal, gpuOutInd, dir, k, 0);
    }
    hipDeviceSynchronize();

    float end_time = timer.elapsedMilliseconds();
    float delta_time = end_time - start_time;

    // Allow one burn-in iteration.
    if (i > 0) {
      stats += benchmark::Stat1_f(delta_time);
    }
  }

  return stats;
}

TEST(TestGpuSelect, test) {
  faiss::gpu::CpuTimer timer;

  printf("B/W batch     n   k     avg (ms)   stddev     throughput\n");
  printf("========================================================\n");
  const int batch_sizes[] = {1, 32, 64};
  const int ns[] = {50000, 500000, 20000000, 2048,32768, 8388608};
  const int ks[] = {10,50,100};
  for (int warp = 0 ; warp <= 1 ; warp++) {
    for (int batch_size : batch_sizes) {
      for(int n : ns) {
        for (int k : ks) {
          bool use_warp = warp == 1;
          auto timings = testForSize(batch_size, n, k, false, use_warp);
          float throughputMBS = (batch_size * n * 4LL + batch_size * k * 8) / 1024 / double(timings.Mean()) * 1000 / 1024;
          printf("%1s %3i %10i %3i %10.3f %10.3f %10.3fMB/s\n", use_warp ? "W" : "B", batch_size, n, k, timings.Mean(), timings.StdDev(), throughputMBS);
        }
      }
    }
  }
}

#if 0
// General test
TEST(TestGpuSelect, test) {
  for (int i = 0; i < 10; ++i) {
    int batch_size = faiss::gpu::randVal(10, 100);
    int n = faiss::gpu::randVal(1, 30000);
    int k = std::min(n, faiss::gpu::randVal(1, 1024));
    bool dir = faiss::gpu::randBool();

    testForSize(batch_size, n, k, dir, false);
  }
}

// Test for k = 1
TEST(TestGpuSelect, test1) {
  for (int i = 0; i < 5; ++i) {
    int batch_size = faiss::gpu::randVal(10, 100);
    int n = faiss::gpu::randVal(1, 30000);
    bool dir = faiss::gpu::randBool();

    testForSize(batch_size, n, 1, dir, false);
  }
}

// Test for where k = #n exactly (we are returning all the values,
// just sorted)
TEST(TestGpuSelect, testExact) {
  for (int i = 0; i < 5; ++i) {
    int batch_size = faiss::gpu::randVal(10, 100);
    int n = faiss::gpu::randVal(1, 1024);
    bool dir = faiss::gpu::randBool();

    testForSize(batch_size, n, n, dir, false);
  }
}

// General test
TEST(TestGpuSelect, testWarp) {
  for (int i = 0; i < 10; ++i) {
    int batch_size = faiss::gpu::randVal(10, 100);
    int n = faiss::gpu::randVal(1, 30000);
    int k = std::min(n, faiss::gpu::randVal(1, 1024));
    bool dir = faiss::gpu::randBool();

    testForSize(batch_size, n, k, dir, true);
  }
}

// Test for k = 1
TEST(TestGpuSelect, test1Warp) {
  for (int i = 0; i < 5; ++i) {
    int batch_size = faiss::gpu::randVal(10, 100);
    int n = faiss::gpu::randVal(1, 30000);
    bool dir = faiss::gpu::randBool();

    testForSize(batch_size, n, 1, dir, true);
  }
}

// Test for where k = #n exactly (we are returning all the values,
// just sorted)
TEST(TestGpuSelect, testExactWarp) {
  for (int i = 0; i < 5; ++i) {
    int batch_size = faiss::gpu::randVal(10, 100);
    int n = faiss::gpu::randVal(1, 1024);
    bool dir = faiss::gpu::randBool();

    testForSize(batch_size, n, n, dir, true);
  }
}
#endif
